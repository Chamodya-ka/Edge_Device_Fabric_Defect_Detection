#include "FeatureComputation.h"
#include "FeatureCalculation.cu"
#include "New_FeatureCalculation.cu"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
GLOBAL void copyArray(float* feat_1,float* feat_2,float* feat_3,float* feat_4,float* feat_5,float* d_features,int N){
	//1D blocks of size 256
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id<N) {

		for (int i = 0 ; i <5 ; i ++){
			if (i==0)
				d_features[id*5 + i ] = feat_1[id];
			if (i==1)
				d_features[id*5 + i ] = feat_2[id];
			if (i==2)
				d_features[id*5 + i ] = feat_3[id];
			if (i==3)
				d_features[id*5 + i ] = feat_4[id];
			if (i==4)
				d_features[id*5 + i ] = feat_5[id];
		}
		
	}
}


void FeatureComputation::getFeatures(float* subGLCM,int gl,int rows, int cols,unsigned int subImgDim,float* features){
    
    //float* d_feat;
    //int* d_subGLCM;
    size_t intsize = sizeof(int);
    size_t floatsize = sizeof(float);
	int blocksX = rows/subImgDim;
	int blocksY = cols/subImgDim;
	int N = blocksX * blocksY;
	std::cout << "N " << N;
	std::cout << " blocksX " << blocksX;
	std::cout << " blocksY " << blocksY << "\n";
    //int N = 64 * 64; // Number of blocks per image
    //int bytes = 4 * gl * gl * N * intsize;
	int num_intermediates = 4 * N;
	// ADDED HERE
	
	

	float* h_feat_1;
	float* h_feat_2;
	float* h_feat_3;
	float* h_feat_4;
	float* h_feat_5;    
	float* d_feat_1;
	float* d_feat_2;
	float* d_feat_3;
	float* d_feat_4;
	float* d_feat_5;
	float* d_inp_1; 
	float* d_inp_2; 
	float* d_inp_3; 
	float* d_inp_4; 
	float* d_inp_5; 
	float* d_features;
	//float* stddX_h;
	//float* stddY_h;
	float* stddX_d;
	float* stddY_d;
	//int* meanX_h;
	//int* meanY_h;
	float* meanX_d;
	float* meanY_d;
  
	h_feat_1 = (float*) malloc(floatsize * N);
	h_feat_2 = (float*) malloc(floatsize * N);
	h_feat_3 = (float*) malloc(floatsize * N);
	h_feat_4 = (float*) malloc(floatsize * N);
	h_feat_5 = (float*) malloc(floatsize * N);
	
	//stddX_h = (float*) malloc(floatsize * num_intermediates);
	//stddY_h = (float*) malloc(floatsize * num_intermediates);
	//meanX_h = (int*) malloc(intsize * num_intermediates);
	//meanY_h = (int*) malloc(intsize * num_intermediates);
	
	hipMalloc(&d_feat_1,floatsize * N);
 	hipMalloc(&d_feat_2,floatsize * N);
	hipMalloc(&d_feat_3,floatsize * N);
	hipMalloc(&d_feat_4,floatsize * N);
	hipMalloc(&d_feat_5,floatsize * N);
	hipMalloc(&d_features,floatsize * N * 5);	
	hipMalloc(&d_inp_1, floatsize * 4 * gl * gl * N);
	hipMalloc(&d_inp_2, floatsize * 4  * gl * gl * N);
	hipMalloc(&d_inp_3, floatsize * 4  * gl * gl * N);
	hipMalloc(&d_inp_4, floatsize * 4  * gl * gl * N);
	hipMalloc(&d_inp_5, floatsize * 4  * gl * gl * N);

	hipMalloc(&stddX_d, floatsize * num_intermediates);	
	hipMalloc(&stddY_d, floatsize * num_intermediates);	
	hipMalloc(&meanX_d, floatsize * num_intermediates);		
	hipMalloc(&meanY_d, floatsize * num_intermediates);		 						
	
	int byte = 4 * gl * gl * N * floatsize;
	float *h_sGLCM;//,*h_sGLCM2,*h_sGLCM3,*h_sGLCM4;
	float *h_sGLCM_1;
	float *h_sGLCM_2;
	float *h_sGLCM_3;
 	float *h_sGLCM_4;
	//hipHostMalloc(&h_sGLCM,byte);
	//hipHostMalloc(&h_sGLCM_1,byte);
	//hipHostMalloc(&h_sGLCM_2,byte);
	//hipHostMalloc(&h_sGLCM_3,byte);
	//hipHostMalloc(&h_sGLCM_4,byte);
	//hipHostMalloc(&)
	
	//float* host_data1= (float*) malloc(byte);
	//float* host_data2= (float*) malloc(byte);
	//float* host_data3= (float*) malloc(byte);	
	//float* host_data4= (float*) malloc(byte);	
	//float* host_data5= (float*) malloc(byte);	
	
	//memcpy(host_data1,subGLCM,byte);
	//memcpy(host_data2,subGLCM,byte);
	//memcpy(host_data2,subGLCM,byte);
	//memcpy(host_data4,subGLCM,byte);
	//memcpy(host_data5,subGLCM,byte);
	//h_sGLCM = host_data1;
	//h_sGLCM_1 = host_data2;
	//h_sGLCM_2 = host_data3;
	//h_sGLCM_3 = host_data4;
	//h_sGLCM_4 = host_data5;  
	
	/*  for (int i = 0 ; i < N * gl *gl * 4 ; i++){
		h_sGLCM[i] = subGLCM[i];
		h_sGLCM_1[i] = subGLCM[i];
		h_sGLCM_2[i] = subGLCM[i];
		h_sGLCM_3[i] = subGLCM[i];
		h_sGLCM_4[i] = subGLCM[i];				
	}  */	

		
	int N_STREAMS = 5; // 5 features
	//float *results[N_STREAMS];
	//int *data[N_STREAMS];	
	
	hipStream_t stream[N_STREAMS];
	for (int i = 0 ; i <N_STREAMS ; i++){
		hipStreamCreate(&stream[i]);
				
	}



 	int THREADS = gl * gl * 2; // HALF THREADS
    int BLOCKS =  N  ;//( gl * gl * 4 * N + THREADS -1 )/ THREADS;
	int copythreads = 256;
	int copyblocks = (N + copythreads -1)/copythreads;
	dim3 intermediateTHREADS(THREADS/2);
	dim3 threads(THREADS);
    dim3 blocks(BLOCKS);
	dim3 cpythreads(copythreads);
	dim3 cpyblocks(copyblocks);
	hipMemsetAsync(stddX_d,0,floatsize *num_intermediates,stream[4]);
	hipMemsetAsync(stddY_d,0,floatsize *num_intermediates,stream[4]);
	hipMemsetAsync(meanX_d,0,floatsize *num_intermediates,stream[4]);
	hipMemsetAsync(meanY_d,0,floatsize *num_intermediates,stream[4]);
	//hipMemcpy(d_inp_1,subGLCM,N * gl * gl *4 * floatsize,hipMemcpyHostToDevice);
	hipMemcpyAsync(d_inp_1,subGLCM,N * gl * gl *4 * floatsize,hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(d_inp_2,subGLCM,N * gl * gl *4 * floatsize,hipMemcpyHostToDevice,stream[1]);
	hipMemcpyAsync(d_inp_3,subGLCM,N * gl * gl *4 * floatsize,hipMemcpyHostToDevice,stream[2]);
	hipMemcpyAsync(d_inp_4,subGLCM,N * gl * gl *4 * floatsize,hipMemcpyHostToDevice,stream[3]);
	hipMemcpyAsync(d_inp_5,subGLCM,N * gl * gl * 4 * floatsize,hipMemcpyHostToDevice,stream[4]);
	
	
	CalculateMeanX<<<blocks,intermediateTHREADS,0,stream[4]>>>(gl, d_inp_1,meanX_d);
	CalculateMeanY<<<blocks,intermediateTHREADS,0,stream[4]>>>(gl, d_inp_1,meanY_d);
	CalculateStddX<<<blocks,intermediateTHREADS,0,stream[4]>>>(gl, d_inp_1,meanX_d,stddX_d);
	CalculateStddY<<<blocks,intermediateTHREADS,0,stream[4]>>>(gl, d_inp_1,meanY_d,stddY_d);
	
	EnergyFeature2<<<blocks,threads,0,stream[0]>>>(gl , d_inp_1 ,d_feat_1);
	ContrastFeature2<<<blocks,threads,0,stream[1]>>>(gl , d_inp_2 ,d_feat_2);
	EntropyFeature2<<<blocks,threads,0,stream[2]>>>(gl , d_inp_3 ,d_feat_3);
	HomogeneityFeature2<<<blocks,threads,0,stream[3]>>>(gl , d_inp_4 ,d_feat_4);
	CorrelationFeature2<<<blocks,threads,0,stream[4]>>>(gl , d_inp_5 ,d_feat_5,meanX_d,meanY_d,stddX_d,stddY_d);



	//hipMemcpyAsync(h_feat_1, d_feat_1, N * floatsize,hipMemcpyDeviceToHost,stream[0]);
	//hipMemcpyAsync(h_feat_2, d_feat_2, N * floatsize,hipMemcpyDeviceToHost,stream[1]);
	//hipMemcpyAsync(h_feat_3, d_feat_3, N * floatsize,hipMemcpyDeviceToHost,stream[2]);
	//hipMemcpyAsync(h_feat_4, d_feat_4, N * floatsize,hipMemcpyDeviceToHost,stream[3]);
	//hipMemcpyAsync(h_feat_5, d_feat_5, N * floatsize,hipMemcpyDeviceToHost,stream[4]);

	gpuErrchk( hipDeviceSynchronize() );
	copyArray<<<cpyblocks,cpythreads>>>(d_feat_1,d_feat_2,d_feat_3,d_feat_4,d_feat_5,d_features,N);
	gpuErrchk( hipDeviceSynchronize() );
	hipFree(d_feat_1);
	hipFree(d_feat_2);
	hipFree(d_feat_3);
	hipFree(d_feat_4);
	hipFree(d_feat_5);
	//hipFree(subGLCM);
	hipFree(d_inp_1);
	hipFree(d_inp_2);
	hipFree(d_inp_3);
	hipFree(d_inp_4);
	hipFree(d_inp_5);
	
	
	
	//free(h_sGLCM);
	//free(h_sGLCM_1);
	///free(h_sGLCM_2);
	//free(h_sGLCM_3);
	//free(h_sGLCM_4);
	//float* h_feat_meanX = (float*) malloc(floatsize * N * 4);
	//float* h_feat_meanY = (float*) malloc(floatsize * N * 4);
	//float* h_feat_stddX = (float*) malloc(floatsize * N * 4);
	//float* h_feat_stddY = (float*) malloc(floatsize * N * 4);
	//hipMemcpyAsync(h_feat_meanX, meanX_d, 4 * N * floatsize,hipMemcpyDeviceToHost,stream[4]);
	//hipMemcpyAsync(h_feat_meanY, meanY_d, 4 * N * floatsize,hipMemcpyDeviceToHost,stream[4]);
	//hipMemcpyAsync(h_feat_stddX, stddX_d, 4 * N * floatsize,hipMemcpyDeviceToHost,stream[4]);
	//hipMemcpyAsync(h_feat_stddY, stddY_d, 4 * N * floatsize,hipMemcpyDeviceToHost,stream[4]);
	hipMemcpy(features, d_features, 5 * N * floatsize,hipMemcpyDeviceToHost);
	//  std::cout << h_feat_1[1493]<<"\n";

/* 	 for (int i = 0 ; i < N ;i++){
		std::cout<<h_feat_1[i]<<" ";
		std::cout<<h_feat_2[i]<<" ";
		std::cout<<h_feat_3[i]<<" ";
		std::cout<<h_feat_4[i]<<" ";
 		std::cout<<h_feat_meanX[i]<<" ";
		std::cout<<h_feat_meanY[i]<<" ";
		std::cout<<h_feat_stddX[i]<<" ";
		std::cout<<h_feat_stddY[i]<<" "; 
		std::cout<<h_feat_5[i]<<" \n";  

	}      */
	
	hipFree(stddX_d);
	hipFree(stddY_d);
	hipFree(meanY_d);
	hipFree(meanX_d);
	// ADDED END HERE
/*

    int THREADS = gl * gl * 4;
    int BLOCKS = ( gl * gl * 4 * N + THREADS -1 )/ THREADS;
    //std::cout<<BLOCKS;
    dim3 threads(THREADS);
    dim3 blocks(BLOCKS);
    
	

    h_feat = (float*)malloc(floatsize * BLOCKS * 5);
    
    hipMalloc(&d_feat, floatsize * 5  * BLOCKS);
    hipMalloc(&d_subGLCM, intsize * gl * gl * 4  * BLOCKS);
    hipMemset(d_feat,0,BLOCKS *  floatsize*5);
    hipMemcpy(d_subGLCM,subGLCM,bytes,hipMemcpyHostToDevice);
    computeFeatures<<<blocks,threads>>>(d_subGLCM,d_feat,gl);
    gpuErrchk( hipPeekAtLastError() );
    //gpuErrchk( hipDeviceSynchronize() );
    hipDeviceSynchronize();
    hipMemcpy(h_feat, d_feat, BLOCKS*5*floatsize, hipMemcpyDeviceToHost );
    hipFree(d_feat);
    hipFree(d_subGLCM);
	*/
     /*for(int h =0 ; h < BLOCKS;h++){
        
        std:: cout << h_feat[h*5 + 0] << " ";
        std:: cout << h_feat[h*5 + 1] << " ";
        std:: cout << h_feat[h*5 + 2] << " ";
        std:: cout << h_feat[h*5 + 3] << " ";
        std:: cout << h_feat[h*5 + 4] << " \n";
        std:: cout << h << "\n";
        //std:: cout << h << " ";
        
    }  */

    return ;



}
