#include "FeatureComputation.h"
#include "FeatureCalculation.cu"
#include "New_FeatureCalculation.cu"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}



float* FeatureComputation::getFeatures(int* subGLCM,int gl,int rows, int cols,unsigned int subImgDim){
    float* h_feat;
    //float* d_feat;
    //int* d_subGLCM;
    size_t intsize = sizeof(int);
    size_t floatsize = sizeof(float);
	int blocksX = rows/subImgDim;
	int blocksY = cols/subImgDim;
	int N = blocksX * blocksY;
	std::cout << "N " << N;
	std::cout << " blocksX " << blocksX;
	std::cout << " blocksY " << blocksY << "\n";
    //int N = 64 * 64; // Number of blocks per image
    //int bytes = 4 * gl * gl * N * intsize;
	int num_intermediates = 4 * N;
	// ADDED HERE
	
	

	float* h_feat_1;
	float* h_feat_2;
	float* h_feat_3;
	float* h_feat_4;
	float* h_feat_5;    
	float* d_feat_1;
	float* d_feat_2;
	float* d_feat_3;
	float* d_feat_4;
	float* d_feat_5;
	int* d_inp_1; 
	int* d_inp_2; 
	int* d_inp_3; 
	int* d_inp_4; 
	int* d_inp_5; 
	//float* stddX_h;
	//float* stddY_h;
	float* stddX_d;
	float* stddY_d;
	//int* meanX_h;
	//int* meanY_h;
	int* meanX_d;
	int* meanY_d;
  
	h_feat_1 = (float*) malloc(floatsize * N);
	h_feat_2 = (float*) malloc(floatsize * N);
	h_feat_3 = (float*) malloc(floatsize * N);
	h_feat_4 = (float*) malloc(floatsize * N);
	h_feat_5 = (float*) malloc(floatsize * N);
	std::cout << " Here1 "<<"\n";	
	//stddX_h = (float*) malloc(floatsize * num_intermediates);
	//stddY_h = (float*) malloc(floatsize * num_intermediates);
	//meanX_h = (int*) malloc(intsize * num_intermediates);
	//meanY_h = (int*) malloc(intsize * num_intermediates);
	
	hipMalloc(&d_feat_1,floatsize * N);
 	hipMalloc(&d_feat_2,floatsize * N);
	hipMalloc(&d_feat_3,floatsize * N);
	hipMalloc(&d_feat_4,floatsize * N);
	hipMalloc(&d_feat_5,floatsize * N);
	std::cout << " Here2 "<<"\n";	
	hipMalloc(&d_inp_1, intsize * 4 * gl * gl * N);
	hipMalloc(&d_inp_2, intsize * 4  * gl * gl * N);
	hipMalloc(&d_inp_3, intsize * 4  * gl * gl * N);
	hipMalloc(&d_inp_4, intsize * 4  * gl * gl * N);
	hipMalloc(&d_inp_5, intsize * 4  * gl * gl * N);

	hipMalloc(&stddX_d, floatsize * num_intermediates);	
	hipMalloc(&stddY_d, floatsize * num_intermediates);	
	hipMalloc(&meanX_d, intsize * num_intermediates);		
	hipMalloc(&meanY_d, intsize * num_intermediates);		 						
	std::cout << " Here3 "<<"\n";
	int byte = 4 * gl * gl * N * intsize;
	float *h_sGLCM;//,*h_sGLCM2,*h_sGLCM3,*h_sGLCM4;
	float *h_sGLCM_1;
	float *h_sGLCM_2;
	float *h_sGLCM_3;
 	float *h_sGLCM_4;
	hipHostMalloc(&h_sGLCM,byte);
	hipHostMalloc(&h_sGLCM_1,byte);
	hipHostMalloc(&h_sGLCM_2,byte);
	hipHostMalloc(&h_sGLCM_3,byte);
	hipHostMalloc(&h_sGLCM_4,byte);
	//hipHostMalloc(&)
	std::cout << " Here4 "<<"\n";
	float* host_data1= (float*) malloc(byte);
	float* host_data2= (float*) malloc(byte);
	float* host_data3= (float*) malloc(byte);	
	float* host_data4= (float*) malloc(byte);	
	float* host_data5= (float*) malloc(byte);	
	std::cout << " Here5 "<<"\n";
	memcpy(host_data1,subGLCM,byte);
	memcpy(host_data2,subGLCM,byte);
	memcpy(host_data2,subGLCM,byte);
	memcpy(host_data4,subGLCM,byte);
	memcpy(host_data5,subGLCM,byte);
	h_sGLCM = host_data1;
	h_sGLCM_1 = host_data2;
	h_sGLCM_2 = host_data3;
	h_sGLCM_3 = host_data4;
	h_sGLCM_4 = host_data5;  
	
	/*  for (int i = 0 ; i < N * gl *gl * 4 ; i++){
		h_sGLCM[i] = subGLCM[i];
		h_sGLCM_1[i] = subGLCM[i];
		h_sGLCM_2[i] = subGLCM[i];
		h_sGLCM_3[i] = subGLCM[i];
		h_sGLCM_4[i] = subGLCM[i];				
	}  */	

		
	int N_STREAMS = 5; // 5 features
	//float *results[N_STREAMS];
	//int *data[N_STREAMS];	
	
	hipStream_t stream[N_STREAMS];
	for (int i = 0 ; i <N_STREAMS ; i++){
		hipStreamCreate(&stream[i]);
				
	}



 	int THREADS = gl * gl * 2; // HALF THREADS
    int BLOCKS =  N  ;//( gl * gl * 4 * N + THREADS -1 )/ THREADS;

	dim3 intermediateTHREADS(THREADS/2);
	dim3 threads(THREADS);
    dim3 blocks(BLOCKS);

	hipMemsetAsync(stddX_d,0,floatsize *num_intermediates,stream[4]);
	hipMemsetAsync(stddY_d,0,floatsize *num_intermediates,stream[4]);
	hipMemsetAsync(meanX_d,0,intsize *num_intermediates,stream[4]);
	hipMemsetAsync(meanY_d,0,intsize *num_intermediates,stream[4]);

	hipMemcpyAsync(d_inp_1,h_sGLCM,N * gl * gl *4 * intsize,hipMemcpyHostToDevice,stream[0]);
	hipMemcpyAsync(d_inp_2,h_sGLCM_1,N * gl * gl *4 * intsize,hipMemcpyHostToDevice,stream[1]);
	hipMemcpyAsync(d_inp_3,h_sGLCM_2,N * gl * gl *4 * intsize,hipMemcpyHostToDevice,stream[2]);
	hipMemcpyAsync(d_inp_4,h_sGLCM_3,N * gl * gl *4 * intsize,hipMemcpyHostToDevice,stream[3]);
	hipMemcpyAsync(d_inp_5,h_sGLCM_4,N * gl * gl * 4 * intsize,hipMemcpyHostToDevice,stream[4]);
	
	
	CalculateMeanX<<<blocks,intermediateTHREADS,0,stream[4]>>>(gl, d_inp_5,meanX_d);
	CalculateMeanY<<<blocks,intermediateTHREADS,0,stream[4]>>>(gl, d_inp_5,meanY_d);
	CalculateStddX<<<blocks,intermediateTHREADS,0,stream[4]>>>(gl, d_inp_5,meanX_d,stddX_d);
	CalculateStddY<<<blocks,intermediateTHREADS,0,stream[4]>>>(gl, d_inp_5,meanY_d,stddY_d);
	
	EnergyFeature2<<<blocks,threads,0,stream[0]>>>(gl , d_inp_1 ,d_feat_1);
	ContrastFeature2<<<blocks,threads,0,stream[1]>>>(gl , d_inp_2 ,d_feat_2);
	EntropyFeature2<<<blocks,threads,0,stream[2]>>>(gl , d_inp_3 ,d_feat_3);
	HomogeneityFeature2<<<blocks,threads,0,stream[3]>>>(gl , d_inp_4 ,d_feat_4);
	CorrelationFeature2<<<blocks,threads,0,stream[4]>>>(gl , d_inp_5 ,d_feat_5,meanX_d,meanY_d,stddX_d,stddY_d);

	gpuErrchk( hipDeviceSynchronize() );
	hipMemcpyAsync(h_feat_1, d_feat_1, N * floatsize,hipMemcpyDeviceToHost,stream[0]);
	hipMemcpyAsync(h_feat_2, d_feat_2, N * floatsize,hipMemcpyDeviceToHost,stream[1]);
	hipMemcpyAsync(h_feat_3, d_feat_3, N * floatsize,hipMemcpyDeviceToHost,stream[2]);
	hipMemcpyAsync(h_feat_4, d_feat_4, N * floatsize,hipMemcpyDeviceToHost,stream[3]);
	hipMemcpyAsync(h_feat_5, d_feat_5, N * floatsize,hipMemcpyDeviceToHost,stream[4]);
	hipFree(d_feat_1);
	hipFree(d_feat_2);
	hipFree(d_feat_3);
	hipFree(d_feat_4);
	hipFree(d_feat_5);
	hipFree(d_inp_1);
	hipFree(d_inp_2);
	hipFree(d_inp_3);
	hipFree(d_inp_4);
	hipFree(d_inp_5);
	hipFree(meanY_d);
	hipFree(meanX_d);
	hipFree(stddX_d);
	hipFree(stddY_d);
	
	//free(h_sGLCM);
	//free(h_sGLCM_1);
	///free(h_sGLCM_2);
	//free(h_sGLCM_3);
	//free(h_sGLCM_4);

	
	std::cout << h_feat_1[1493]<<"\n";	
	for (int i = 0 ; i < N ;i++){
		std::cout<<h_feat_1[i]<<" ";
		std::cout<<h_feat_2[i]<<" ";
		std::cout<<h_feat_3[i]<<" ";
		std::cout<<h_feat_4[i]<<" ";
		std::cout<<h_feat_5[i]<<" \n";  
	}   
	
	// ADDED END HERE
/*

    int THREADS = gl * gl * 4;
    int BLOCKS = ( gl * gl * 4 * N + THREADS -1 )/ THREADS;
    //std::cout<<BLOCKS;
    dim3 threads(THREADS);
    dim3 blocks(BLOCKS);
    
	

    h_feat = (float*)malloc(floatsize * BLOCKS * 5);
    
    hipMalloc(&d_feat, floatsize * 5  * BLOCKS);
    hipMalloc(&d_subGLCM, intsize * gl * gl * 4  * BLOCKS);
    hipMemset(d_feat,0,BLOCKS *  floatsize*5);
    hipMemcpy(d_subGLCM,subGLCM,bytes,hipMemcpyHostToDevice);
    computeFeatures<<<blocks,threads>>>(d_subGLCM,d_feat,gl);
    gpuErrchk( hipPeekAtLastError() );
    //gpuErrchk( hipDeviceSynchronize() );
    hipDeviceSynchronize();
    hipMemcpy(h_feat, d_feat, BLOCKS*5*floatsize, hipMemcpyDeviceToHost );
    hipFree(d_feat);
    hipFree(d_subGLCM);
	*/
     /*for(int h =0 ; h < BLOCKS;h++){
        
        std:: cout << h_feat[h*5 + 0] << " ";
        std:: cout << h_feat[h*5 + 1] << " ";
        std:: cout << h_feat[h*5 + 2] << " ";
        std:: cout << h_feat[h*5 + 3] << " ";
        std:: cout << h_feat[h*5 + 4] << " \n";
        std:: cout << h << "\n";
        //std:: cout << h << " ";
        
    }  */

    return h_feat;



}
