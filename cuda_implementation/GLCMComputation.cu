#include "hip/hip_runtime.h"
#include "GLCMComputation.h"
#include <assert.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

 __device__ void warpReduceGLCM(volatile float* sdata, unsigned int tid) {
    sdata[tid] += sdata[tid + 32]; 
    sdata[tid] += sdata[tid + 16]; 
    sdata[tid] += sdata[tid +  8];
    sdata[tid] += sdata[tid +  4]; 
    sdata[tid] += sdata[tid +  2]; 
    sdata[tid] += sdata[tid +  1];  
} 
__device__ void normalizeGLCM(volatile int* histogramSubGLCM,volatile float* subGLCM,unsigned  int id,int gl){
    __shared__ float tempGLCM[8*8*4];
    /*   for (unsigned int i =0 ;i < 4; i++){
        tempGLCM[id+gl*gl*i] = histogramSubGLCM[id+gl*gl*i];
    }  */ 
    
    subGLCM[id] = histogramSubGLCM[id];
    

     tempGLCM[id] = histogramSubGLCM[id];
    __syncthreads();
    /* for (unsigned int s=gl*gl/2; s>32; s>>=1) {
        if (id < s) {
            tempGLCM[id]         += tempGLCM[id + s];
            tempGLCM[id+gl*gl*1] += tempGLCM[id+gl*gl*1 + s];
            tempGLCM[id+gl*gl*2] += tempGLCM[id+gl*gl*2 + s];
            tempGLCM[id+gl*gl*3] += tempGLCM[id+gl*gl*3 + s];
        }
        __syncthreads();
    } */
    if (id < 32){
        warpReduceGLCM(tempGLCM, id);
    }
    
    else if (64<=id <96){
        warpReduceGLCM(tempGLCM, id);
    }else if(128<=id<160){
        warpReduceGLCM(tempGLCM, id);
    }else if(192<=id<224){
        warpReduceGLCM(tempGLCM, id);
    } 
    __syncthreads();
    subGLCM[id] = __fdividef(histogramSubGLCM[id],tempGLCM[gl*gl*(id/(gl*gl))]);
    
    /* subGLCM[id + gl * gl * 0] = __fdividef(subGLCM[id + gl * gl * 0],tempGLCM[gl*gl*0]);

    subGLCM[id + gl * gl * 1] = 6.0;//tempGLCM[gl*gl*1];//__fdividef(subGLCM[id + gl * gl * 1],tempGLCM[gl*gl*1]);

    subGLCM[id + gl * gl * 2] = tempGLCM[gl*gl*2];//__fdividef(subGLCM[id + gl * gl * 2],tempGLCM[gl*gl*2]);

    subGLCM[id + gl * gl * 3] = tempGLCM[gl*gl*3];//__fdividef(subGLCM[id + gl * gl * 3],tempGLCM[gl*gl*3]);
 */

}

GLOBAL void ComputeCoOccurenceMat(const int *pixels, float *d_out, const int N,const int rows, const int cols
            , int gl,int sizeDout){
                //float* feature; 
                __shared__ float subGLCM[8 * 8 * 4];
                __shared__ int subHistogrambGLCM[8 * 8 * 4];
                //__shared__ float featureVector[5];
                int blockID = blockIdx.x + blockIdx.y *gridDim.x;
                int idX = blockID * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
                int localIdX = threadIdx.x + threadIdx.y * blockDim.x;
                if (localIdX<gl*gl*4){
                    subHistogrambGLCM[localIdX] = 0;
                }
                
                __syncthreads();
                
                if (idX< N){ 
                    if(idX + 1 < N && floorf((idX + 1)/blockDim.x)==floorf(idX/blockDim.x)){
                        //d = 0 - Compare and add Current index and Current Index  + 1  
                        atomicAdd( &subHistogrambGLCM[pixels[idX] * gl + pixels[idX+1] ],1);   
						//atomicAdd( &d_out[blockID * gl * gl * 4 + (int)pixels[idX] * gl + (int)pixels[idX+1]] , (int)1);
                    }
                    if(((int(idX)-int(blockDim.x))>=0) && (floorf((idX - blockDim.x)/(blockDim.x * blockDim.y))== floorf(idX /(blockDim.x * blockDim.y)))){
                        //d = 90
                        atomicAdd( &subHistogrambGLCM[(1 * gl * gl) + pixels[idX] * gl +  pixels[idX-blockDim.x]], 1);
						//atomicAdd( &d_out[blockID * gl * gl * 4 + gl*gl*1 +(int)pixels[idX] * gl + (int)pixels[idX+1]] , (int)1);
                    }
                    if (floorf((idX - blockDim.x+1)/(blockDim.x * blockDim.y) )== floorf(idX /(blockDim.x * blockDim.y))){
                        //d = 45
                        if (floorf((idX - blockDim.x+1)/blockDim.x)  < floorf(idX /blockDim.x))
                            atomicAdd( &subHistogrambGLCM[(2 * gl * gl) + pixels[idX] * gl +  pixels[idX-blockDim.x+1]], 1);
							//atomicAdd( &d_out[blockID * gl * gl * 4 + gl*gl*2 +(int)pixels[idX] * gl + (int)pixels[idX+1]] ,(int) 1);
                    }
                    if(floorf((idX - blockDim.x-1)/(blockDim.x * blockDim.y))== floorf(idX /(blockDim.x * blockDim.y))){
                        //d = 135
                        if (floorf((idX - blockDim.x-1)/blockDim.x) + 1 == floorf(idX /blockDim.x)){
                            atomicAdd( &subHistogrambGLCM[(3 * gl* gl)  + pixels[idX] * gl] +  pixels[idX - blockDim.x-1], 1);
							//atomicAdd( &d_out[blockID * gl * gl * 4 + gl*gl*3 +(int)pixels[idX] * gl + (int)pixels[idX+1]] , (int)1);
                        }                  
                    }
                }
                
                 __syncthreads();
                //used to normalize the GLCM
                /* if(localIdX < gl*gl){
                    normalizeGLCM(subGLCM,localIdX,gl);
                }  */
                 if (localIdX<gl*gl*4){
                    normalizeGLCM(subHistogrambGLCM,subGLCM,localIdX,gl);
                }  
                __syncthreads();
                if (localIdX< gl * gl * 4){
                    if (blockID * gl * gl * 4 + localIdX < sizeDout)
                    	d_out[blockID * gl * gl * 4 + localIdX] = subGLCM[localIdX];
                    
                }            
                 
            }


float* GLCMComputation :: GetSubGLCM(Image img,const int d, const int angle,unsigned int subImgDim){
    float* h_out;
    //float* h_feat;
    int* d_pixels;
    //int* d_out;
    float* d_out;
    //float* d_feat;
    std::vector<int> v = img.getPixels();
    int* host_pixels = &v[0];
    int rows = img.get_rows();
    int cols = img.get_cols();
    int gl = img.get_maxGL() + 1;
    int N = rows * cols;

    size_t bytes = rows * cols * sizeof(int);
    size_t intsize = sizeof(int);
    size_t floatsize = sizeof(float);
   //size_t floatsize = sizeof(float);
    

    int THREADS = subImgDim;
    //rows = cols because square shaped
    int BLOCKSx = ( rows + THREADS -1 )/ THREADS;
    int BLOCKSy = ( cols + THREADS -1 )/ THREADS;

	//int sizeDout = BLOCKSx*BLOCKSy*gl*gl*intsize*4;
    int sizeDout = BLOCKSx*BLOCKSy*gl*gl*floatsize*4;
    std::cout<<"BLOCKSx :";
    std::cout<< BLOCKSx << endl;
    std::cout<<"BLOCKSy :";
    std::cout<< BLOCKSy << endl;
    dim3 threadsPerBlock(THREADS,THREADS,1);
    dim3 blocksPerGrid(BLOCKSx,BLOCKSy,1);

    h_out = (float*)malloc(sizeDout);
    std::cout<<"TESTING!@";
    //h_feat = (float*)malloc(floatsize * BLOCKS * BLOCKS * 5);

    hipMalloc(&d_pixels,bytes);
    hipMalloc(&d_out,sizeDout);
    hipMemset(d_out, 0,sizeDout);
    //hipMalloc(&d_feat,BLOCKS * BLOCKS * floatsize*5);
    
    //hipMemset(d_feat,0,BLOCKS * BLOCKS * floatsize*5);
    hipMemcpy(d_pixels,host_pixels,bytes,hipMemcpyHostToDevice);
    
    //LAUCNCH KERNEL HERE
    ComputeCoOccurenceMat<<<blocksPerGrid,threadsPerBlock>>>(d_pixels,d_out,N,rows,cols,8,sizeDout);
    gpuErrchk( hipDeviceSynchronize() );
    
    hipMemcpy(h_out, d_out, BLOCKSx*BLOCKSy*gl*gl*floatsize*4, hipMemcpyDeviceToHost );

    //hipMemcpy(h_feat, d_feat, BLOCKS*BLOCKS*floatsize*5, hipMemcpyDeviceToHost );
    
    hipFree(d_pixels);
    hipFree(d_out);
    //hipFree(d_feat);


    return h_out;
}

