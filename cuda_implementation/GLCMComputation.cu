#include "hip/hip_runtime.h"
#include "GLCMComputation.h"
#include <assert.h>
#include "FeatureCalculation.cu"

/* DEV void EnergyFeature(int id, int gl, int* subGLCM, float* feature){
    __shared__ float a,b,c,d;
    switch (id)
    {
    case 0:
        a=0;
        break;
    case 1:
        b=0;
    break;
    case 2:
        c=0;
    break;
    case 3:
        d=0;
    break;
    
    default:
        break;
    }

    __syncthreads();
    
    if (id<gl*gl){
        atomicAdd(&a,pow(subGLCM[id],2));
    }
    else if(id<gl*gl*2){
        atomicAdd(&b,pow(subGLCM[id],2));
    }
    else if(id<gl*gl*3){
        atomicAdd(&c,pow(subGLCM[id],2));
    }
    else if(id<gl*gl*4){
        atomicAdd(&d,pow(subGLCM[id],2));
    }
    //printf("%d",a);
    __syncthreads();
    if (id<=0)
        feature[id] = (float)(a+b+c+d)/4;

} */

GLOBAL void ComputeCoOccurenceMat(const int *pixels, int *d_out, float *d_feat, const int N,const int rows, const int cols
            , int gl){
                float* feature; 
                int* subMat;
                //HERE HARDCORED SIZE OF GL * GL * 4 DUE TO CONSTANT INT REQUIREMENT
                __shared__ int subGLCM[8 * 8 * 4];
                __shared__ float featureVector[5];
                feature = featureVector;
                subMat = subGLCM;
                //int row = blockIdx.x * blockDim.x + threadIdx.x;
                //int col = blockIdx.y * blockDim.y + threadIdx.y;
                //int idX = col + row * cols;
                int blockID = blockIdx.x + blockIdx.y *gridDim.x;
                int idX = blockID * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
                int localIdX = threadIdx.x + threadIdx.y * blockDim.x;
                if (localIdX<gl*gl*4){
                    subGLCM[localIdX] = 0;
                    
                }
                
                __syncthreads();
                
                if (idX< N){ 
                    if(idX + 1 < N && floorf((idX + 1)/blockDim.x)==floorf(idX/blockDim.x)){
                        //d = 0 - Compare and add Current index and Current Index  + 1  
                        atomicAdd( &subGLCM[pixels[idX] * gl + pixels[idX+1] ],1);   
                    }
                    if(((int(idX)-int(blockDim.x))>=0) && (floorf((idX - blockDim.x)/(blockDim.x * blockDim.y))== floorf(idX /(blockDim.x * blockDim.y)))){
                        //d = 90
                        atomicAdd( &subGLCM[(1 * gl * gl) + pixels[idX] * gl +  pixels[idX-blockDim.x]], 1);
                    }
                    if (floorf((idX - blockDim.x+1)/(blockDim.x * blockDim.y) )== floorf(idX /(blockDim.x * blockDim.y))){
                        //d = 45
                        if (floorf((idX - blockDim.x+1)/blockDim.x)  < floorf(idX /blockDim.x))
                            atomicAdd( &subGLCM[(2 * gl * gl) + pixels[idX] * gl +  pixels[idX-blockDim.x+1]], 1);
                    }
                    if(floorf((idX - blockDim.x-1)/(blockDim.x * blockDim.y))== floorf(idX /(blockDim.x * blockDim.y))){
                        //d = 135
                        if (floorf((idX - blockDim.x-1)/blockDim.x) + 1 == floorf(idX /blockDim.x)){
                            atomicAdd( &subGLCM[(3 * gl* gl)  + pixels[idX] * gl] +  pixels[idX - blockDim.x-1], 1);
                        }                  
                    }
                }
                
                __syncthreads();
                
                
                if (localIdX< gl * gl * 4){
                    
                    //COMMENTED TO TEST CONTRAST
                     /* EnergyFeature(localIdX,gl,subMat,feature);
                    __syncthreads();
                    if (localIdX<=0){
                        //printf("%f",feature[localIdX]);
                        d_feat[blockID + 0] = (float)featureVector[localIdX];
                    }  */

                    ContrastFeature(localIdX,gl,subMat,feature);
                    __syncthreads();
                    if (localIdX==1){
                        //printf("%f",feature[localIdX]);
                        d_feat[blockID + 0] = (float)featureVector[localIdX];
                    }
                    //printf("%d\f",&feature); // LOOKS WRONG CHECK
                    d_out[(blockIdx.x + blockIdx.y * gridDim.x) * gl * gl * 4 + localIdX] = subGLCM[localIdX];
                    
                }
                
                

                 
            }


int* GLCMComputation :: GetSubGLCM(Image img,const int d, const int angle){
    int* h_out;
    float* h_feat;
    int* d_pixels;
    int* d_out;
    float* d_feat;
    std::vector<int> v = img.getPixels();
    int* host_pixels = &v[0];
    int rows = img.get_rows();
    int cols = img.get_cols();
    int gl = img.get_maxGL();
    int N = rows * cols;

    size_t bytes = rows * cols * sizeof(int);
    size_t intsize = sizeof(int);
    size_t floatsize = sizeof(float);
    

    int THREADS = 32;
    //rows = cols because square shaped
    int BLOCKS = ( rows + THREADS -1 )/ THREADS;
    cout<<"BLOCKS :";
    cout<< BLOCKS << endl;
    dim3 threadsPerBlock(THREADS,THREADS);
    dim3 blocksPerGrid(BLOCKS,BLOCKS);

    h_out = (int*)malloc(intsize * gl *gl *4 *BLOCKS *BLOCKS);
    h_feat = (float*)malloc(floatsize * BLOCKS * BLOCKS * 5);

    hipMalloc(&d_pixels,bytes);
    hipMalloc(&d_out,BLOCKS*BLOCKS*gl*gl*intsize*4);
    hipMalloc(&d_feat,BLOCKS * BLOCKS * floatsize*5);
    hipMemset(d_out, 0,BLOCKS*BLOCKS*gl*gl*intsize*4);
    hipMemset(d_feat,0,BLOCKS * BLOCKS * floatsize*5);
    hipMemcpy(d_pixels,host_pixels,bytes,hipMemcpyHostToDevice);
    
    //LAUCNCH KERNEL HERE
    ComputeCoOccurenceMat<<<blocksPerGrid,threadsPerBlock>>>(d_pixels,d_out,d_feat,N,rows,cols,8);
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, BLOCKS*BLOCKS*gl*gl*intsize*4, hipMemcpyDeviceToHost );
    hipMemcpy(h_feat, d_feat, BLOCKS*BLOCKS*floatsize*5, hipMemcpyDeviceToHost );
    
    hipFree(d_pixels);
    hipFree(d_out);
    hipFree(d_feat);
     for(int h =0 ; h < BLOCKS*BLOCKS*5;h++){
        cout<< ("%f",h_feat[h])<< " ";
    } 

    return h_out;
}

